#include "hip/hip_runtime.h"
#include "rmsd.cuh"
#include <algorithm>
#include <stdio.h>

double compute_rmsd_gpu(std::vector<point*> s1, std::vector<point*> s2) {
	//first convert the vectors into arrays
	int size1 = s1.size();
	int size2 = s2.size();

	int* h_s1Ax = (int*)malloc(sizeof(point) * size1);
	int* h_s1Ay = (int*)malloc(sizeof(point) * size1);
	int* h_s1Az = (int*)malloc(sizeof(point) * size1);

	int* h_s2Ax = (int*)malloc(sizeof(point) * size2);
	int* h_s2Ay = (int*)malloc(sizeof(point) * size2);
	int* h_s2Az = (int*)malloc(sizeof(point) * size2);

	//allocate the host while copying arrays
	int i;
	point* p;
	for (i = 0; i < size1; i++) {
		p = s1.at(i);
		h_s1Ax[i] = p->x;
		h_s1Ay[i] = p->y;
		h_s1Az[i] = p->z;
	}
	for (i = 0; i < size2; i++) {
		p = s2.at(i);
		h_s2Ax[i] = p->x;
		h_s2Ay[i] = p->y;
		h_s2Az[i] = p->z;
	}

	//allocate space on the device
	int* d_s1Ax = 0;
	int* d_s1Ay = 0;
	int* d_s1Az = 0;
	
	int* d_s2Ax = 0;
	int* d_s2Ay = 0;
	int* d_s2Az = 0;
	
	hipMalloc(&d_s1Ax, sizeof(int) * size1);
	hipMalloc(&d_s1Ay, sizeof(int) * size1);
	hipMalloc(&d_s1Az, sizeof(int) * size1);
	
	hipMalloc(&d_s2Ax, sizeof(int) * size2);
	hipMalloc(&d_s2Ay, sizeof(int) * size2);
	hipMalloc(&d_s2Az, sizeof(int) * size2);

	//allocate a results array
	int* d_sR = 0;
	hipMalloc(&d_sR, sizeof(int) * std::min(size1, size2));

	//copy memory
	hipMemcpy(d_s1Ax, h_s1Ax, sizeof(int) * size1, hipMemcpyHostToDevice);
	hipMemcpy(d_s1Ay, h_s1Ay, sizeof(int) * size1, hipMemcpyHostToDevice);
	hipMemcpy(d_s1Az, h_s1Az, sizeof(int) * size1, hipMemcpyHostToDevice);

	hipMemcpy(d_s2Ax, h_s2Ax, sizeof(int) * size2, hipMemcpyHostToDevice);
	hipMemcpy(d_s2Ay, h_s2Ay, sizeof(int) * size2, hipMemcpyHostToDevice);
	hipMemcpy(d_s2Az, h_s2Az, sizeof(int) * size2, hipMemcpyHostToDevice);

	//now we can start the computation
	double n_inverse = 1.0 / std::min(size1, size2);
	int sum = 0;
	int blocksPerGrid = 2;
	int threadsPerBlock = 1000;
	sumPointDistancesGPU<<<blocksPerGrid, threadsPerBlock>>>(d_s1Ax, d_s1Ay, d_s1Az, d_s2Ax, d_s2Ay, d_s2Az, d_sR);
	hipDeviceSynchronize();

	int* results = (int*)malloc(sizeof(int) * std::min(size1, size2));
	hipMemcpy(results, d_sR, std::min(size1, size2) * sizeof(int), hipMemcpyDeviceToHost);

	#pragma omp parallel for reduction(+:sum)
	for (i = 0; i < std::min(size1, size2); i++) {
		sum += results[i];
	}

	double radicand = n_inverse * double(sum);
	return sqrt(radicand);
}

__global__
void sumPointDistancesGPU(int* s1x, int* s1y, int* s1z, int* s2x, int* s2y, int* s2z, int *r) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int x1 = s1x[i];
	int y1 = s1y[i];
	int z1 = s1z[i];
	int x2 = s2x[i];
	int y2 = s2y[i];
	int z2 = s2z[i];

	int distance = (x1 - x2)*(x1 - x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2);
	//printf("distance between (%d, %d, %d) and (%d, %d, %d): %d\n", x1, y1, z1, x2, y2, z2, distance);
	r[i] = distance;
}